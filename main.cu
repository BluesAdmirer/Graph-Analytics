#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <queue>
#include <algorithm>
#include <fstream>
#include "main.h"

using namespace std;

int inc=0;

int find(int u,int parent[]){
	if(parent[u]<0) return u;
	return parent[u]=find(parent[u],parent);
}

int unionit(int u,int v,int parent[]){
	int pu=find(u,parent);
	int pv=find(v,parent);
	if(pu==pv) return 0;
	if(-parent[pu]>-parent[pv]){   
		parent[pu]=parent[pu]+parent[pv];
		parent[pv]=pu;
	}   
	else{   
		parent[pv]=parent[pu]+parent[pv];
		parent[pu]=pv;
	}   
	return 1;
}

void dfs(vector < vector < int > > & graph,int visit[],int nvisit[],int node){
	nvisit[node]=1;
	for(int i=0;i<graph[node].size();i++)
		if(nvisit[graph[node][i]]==-1)
			dfs(graph,visit,nvisit,graph[node][i]);
	visit[inc++]=node;
}

void rdfs(vector < vector < int > > & graph,int nvisit[],int node,int component[],int com){
	nvisit[node]=1;
	component[node]=com;
	for(int i=0;i<graph[node].size();i++)
		if(nvisit[graph[node][i]]==-1)
			rdfs(graph,nvisit,graph[node][i],component,com);
}

void topobfs(vector < vector < int > > & graph, int order[], int visit[]){
	int i,j;
	queue < int > line;
	memset(visit, -1, sizeof(int)*graph.size());
	int indegree[graph.size()];
	memset(indegree,0,sizeof(indegree));
	for(i=0;i<graph.size();i++){
		for(j=0;j<graph[i].size();j++){
			indegree[graph[i][j]]++;
		}
	}
	for(i=0;i<graph.size();i++){
		if(!indegree[i]) {
			line.push(i);
			visit[i]=0;
			indegree[i]--;
		}
	}
	while(!line.empty()){
		int node=line.front();
		line.pop();
		order[inc++]=node;
		for(i=0;i<graph[node].size();i++)
		{
			indegree[graph[node][i]]--;
			if(indegree[graph[node][i]]==0)
			{
				line.push(graph[node][i]);
				visit[graph[node][i]]=visit[node]+1;
			}
		}
	}
}

int optchain=0, optdead=0, optident=0;

int main(){

	int w;

	ifstream fin;
	fin.open("input.txt");
	
	ofstream fout;
	fout.open("output.txt");
	
	int n,m;
	fin >> n >> m;
	
	int i,j,u,v;
	vector < vector < int > > graph(n), rgraph(n), rcgraph(n), rcwgraph(n);

	int outdeg[n];
	memset(outdeg,0,sizeof(outdeg));

	for(i=0;i<m;i++){
		fin >> u >> v, --u,--v;
		graph[u].push_back(v);
		rgraph[v].push_back(u);
		outdeg[u]++;
	}

	int visit[n];
	memset(visit, -1, sizeof(visit));

	int component[n];
	memset(component, -1, sizeof(component));
	
	int nvisit[n];
	memset(nvisit, -1, sizeof(nvisit));
	
	for(i=0;i<n;i++){
		if(nvisit[i]==-1) {
			dfs(graph,visit,nvisit,i);	
		}
	}

	memset(nvisit,-1,sizeof(nvisit));
	
	int com=0;
	for(i=n-1;i>=0;i--){
		if(nvisit[visit[i]]==-1){
			rdfs(rgraph,nvisit,visit[i],component,com);
			com++;
		}
	}

	for(i=0;i<n;i++){
		for(j=0;j<rgraph[i].size();j++){
			if(component[i]==component[rgraph[i][j]]){ 
				rcgraph[i].push_back(rgraph[i][j]);
			}
			else{ 
				rcwgraph[i].push_back(rgraph[i][j]);
			}
		}
	}
	
	vector < vector < int > > members(com), compgr(com);
	
	for(i=0;i<n;i++){
		for(j=0;j<graph[i].size();j++){
			if(component[i]!=component[graph[i][j]]){
				compgr[component[i]].push_back(component[graph[i][j]]);
			}
		}
	}

	int order[com];
	memset(nvisit,0,sizeof(nvisit));
	
	inc=0;
	topobfs(compgr,order,nvisit);
	
	int number[n];
	memset(number,0,sizeof(number));

	for(i=0;i<n;i++){
		if(rgraph[i].size()==1){
			number[rgraph[i][0]]++;
		}
	}

	int equiperc=0;
	for(i=0;i<n;i++){
		equiperc=equiperc+max(0,number[i]-1);
	}
	
	double vai=double(equiperc)/n;
	double ratio=double(m)/n;

	if(vai>0.06 && ratio>3.0)
		optident=1;
	
	int parent2[n];
	memset(parent2,-1,sizeof(parent2));
	
	int parent1[n];
	memset(parent1,-1,sizeof(parent1));
	
	for(i=0;i<n;i++){
		if(rgraph[i].size()>1 || graph[i].size()>1 ) 
			continue;
		for(j=0;j<rcgraph[i].size();j++){
			if(graph[rcgraph[i][j]].size()>1 || rgraph[rcgraph[i][j]].size()>1) 
				continue;
			if(unionit(rcgraph[i][j],i,parent1)){
				parent2[i]=rcgraph[i][j];
			}
		}
	}
	
	int redir[n];
	for(i=0;i<n;i++){
		redir[i]=i;
	}

	int levelz[n];
	memset(levelz,0,sizeof(levelz));
	
	double randomp=0.15/n;

	double powers[n];
	powers[0]=1;
	for(i=1;i<n;i++){
		powers[i]=powers[i-1]*0.85;
	}
	
	int vac=0;
	
	for(i=0;i<n;i++)
	{
		if(rgraph[i].size()>1 || graph[i].size()>1 ) continue;
		if(parent2[i]!=-1) continue;
		int node=i;
		int iterations=0;
		while(graph[node].size())
		{
			node=graph[node][0];
			if(component[node]!=component[i] || node==i || graph[node].size()>1 || rgraph[node].size()>1) break;
			iterations++;
			redir[node]=i;
			levelz[node]=iterations;
		}
		vac=vac+iterations;
	}

	double rac=double(vac)/n;
	if(rac>0.2)
		optchain=1;

	// optdead =  1;

	if(optident==1 && optchain==0 && optdead==0){
		int parent[n];
		vector < vector < int > > left(com);
		for(i=0;i<n;i++){
			parent[i]=i;
		}
		vector < vector <  pair  <  pair < long long , int > , int >  > > hvalues(n);
		for(i=0;i<n;i++){
			if(rgraph[i].size()!=1 && rgraph[i].size()!=2) 
				continue;
			if(rgraph[i].size()==1){
				hvalues[(rgraph[i][0])%n].push_back(make_pair(make_pair(rgraph[i][0],component[i]),i));
			}
			else{
				long long val=max(rgraph[i][1]+1,rgraph[i][0]+1)*(long long)(n+1)+min(rgraph[i][0]+1,rgraph[i][1]+1);
				hvalues[(val)%(long long)n ].push_back(make_pair(make_pair(val,component[i]),i));
			}
		}
		for(i=0;i<n;i++){
			sort(hvalues[i].begin(),hvalues[i].end());
		}
		for(int k=0;k<n;k++)
		{
			for(i=0;i<hvalues[k].size();i++)
			{
				for(j=i;j<hvalues[k].size() && hvalues[k][j].first==hvalues[k][i].first ;j++)
				{
					parent[hvalues[k][j].second]=hvalues[k][i].second;
				}
				i=j-1;
			}
		}
		hvalues.clear();
		int noo=0;
		for(i=0;i<n;i++){
			if(parent[i]==i) {
				members[component[i]].push_back(i);
			}
			else{
				left[component[i]].push_back(i);
				noo++;
			}
		}

		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}
		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		int thresh=100000;

		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<par.size()-1;i++){
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++){
				int sum=0;
				for(j=0;j<members[order[w]].size();j++){
					sum=sum+rgraph[members[order[w]][j]].size();
				}
				if(sum>thresh){
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++){
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					}
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++){
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					}
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
				long long val=computeparalleli(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial);
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++){
				long long val=computeranki(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){ 
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}

	if(optident==1 && optchain==0 && optdead==1)	
	{
		int parent[n];
		vector < vector < int > > left(com);
		for(i=0;i<n;i++){
			parent[i]=i;
		}
		vector < vector <  pair  <  pair < long long , int > , int >  > > hvalues(n);
		for(i=0;i<n;i++){
			if(rgraph[i].size()!=1 && rgraph[i].size()!=2) 
				continue;
			if(rgraph[i].size()==1){
				hvalues[(rgraph[i][0])%n].push_back(make_pair(make_pair(rgraph[i][0],component[i]),i));
			}
			else{
				long long val=max(rgraph[i][1]+1,rgraph[i][0]+1)*(long long)(n+1)+min(rgraph[i][0]+1,rgraph[i][1]+1);
				hvalues[(val)%(long long)n ].push_back(make_pair(make_pair(val,component[i]),i));
			}
		}
		for(i=0;i<n;i++){
			sort(hvalues[i].begin(),hvalues[i].end());
		}
		for(int k=0;k<n;k++){
			for(i=0;i<hvalues[k].size();i++){
				for(j=i;j<hvalues[k].size() && hvalues[k][j].first==hvalues[k][i].first ;j++){
					parent[hvalues[k][j].second]=hvalues[k][i].second;
				}
				i=j-1;
			}
		}
		hvalues.clear();
		int noo=0;
		for(i=0;i<n;i++){
			if(parent[i]==i){
				members[component[i]].push_back(i);
			}
			else{
				left[component[i]].push_back(i);
				noo++;
			}
		}
		
		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}

		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		int thresh=100000;
		
		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<par.size()-1;i++){
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++){
				int sum=0;
				for(j=0;j<members[order[w]].size();j++){
					sum=sum+rgraph[members[order[w]][j]].size();
				}
				if(sum>thresh){
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
				long long val=computeparallelid(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial);
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
				long long val=computerankid(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial);
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}
	
	if(optident==0 && optchain==0 && optdead==0)
	{
		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}

		vector < int > par;
		par.push_back(0);

		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		int thresh=100000;
		
		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<n;i++){
			members[component[i]].push_back(i);
		}

		for(i=0;i<par.size()-1;i++)
		{
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++)
			{
				int sum=0;
				for(j=0;j<members[order[w]].size();j++){
					sum=sum+rgraph[members[order[w]][j]].size();
				}
				if(sum>thresh){
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
			{
				long long val=computeparallel(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial);
			}
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
			{
				long long val=computerank(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}

	if(optident==0 && optchain==0 && optdead==1)
	{
		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}

		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		int thresh=100000;
		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<n;i++){
			members[component[i]].push_back(i);
		}

		for(i=0;i<par.size()-1;i++)
		{
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++)
			{
				int sum=0;
				for(j=0;j<members[order[w]].size();j++)
					sum=sum+rgraph[members[order[w]][j]].size();
				if(sum>thresh){
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
			{
				long long val=computeparalleld(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial);
			}
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
			{
				long long val=computerankd(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}

	if(optident==0 && optchain==1 && optdead==0)
	{
		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}

		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		for(i=0;i<n;i++){
			members[component[i]].push_back(i);
		}

		int thresh=100000;
		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<par.size()-1;i++)
		{
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++)
			{
				int sum=0;
				for(j=0;j<members[order[w]].size();j++){
					sum=sum+rgraph[members[order[w]][j]].size();
				}
				if(sum>thresh)
				{
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
			{
				long long val=computeparallelc(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
			{
				long long val=computerankc(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}

	if(optident==0 && optchain==1 && optdead==1)
	{
		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}

		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		for(i=0;i<n;i++){
			members[component[i]].push_back(i);
		}

		int thresh=100000;
		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<par.size()-1;i++)
		{
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++)
			{
				int sum=0;
				for(j=0;j<members[order[w]].size();j++){
					sum=sum+rgraph[members[order[w]][j]].size();
				}
				if(sum>thresh)
				{
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
			{
				long long val=computeparalleldc(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
			{
				long long val=computerankdc(rcgraph,members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}
	
	if(optident==1 && optchain==1 && optdead==0)
	{
		int parent[n];
		vector < vector < int > > left(com);

		for(i=0;i<n;i++){
			parent[i]=i;
		}

		vector < vector <  pair  <  pair < long long , int > , int >  > > hvalues(n);

		for(i=0;i<n;i++){
			if(rgraph[i].size()!=1 && rgraph[i].size()!=2) 
				continue;
			if(rgraph[i].size()==1){
				hvalues[(rgraph[i][0])%n].push_back(make_pair(make_pair(rgraph[i][0],component[i]),i));
			}
			else{
				long long val=max(rgraph[i][1]+1,rgraph[i][0]+1)*(long long)(n+1)+min(rgraph[i][0]+1,rgraph[i][1]+1);
				hvalues[(val)%(long long)n ].push_back(make_pair(make_pair(val,component[i]),i));
			}
		}

		for(i=0;i<n;i++){
			sort(hvalues[i].begin(),hvalues[i].end());
		}

		for(int k=0;k<n;k++){
			for(i=0;i<hvalues[k].size();i++){
				for(j=i;j<hvalues[k].size() && hvalues[k][j].first==hvalues[k][i].first ;j++){
					parent[hvalues[k][j].second]=hvalues[k][i].second;
				}
				i=j-1;
			}
		}

		hvalues.clear();
		int noo=0;
		for(i=0;i<n;i++){
			if(parent[i]==i) 
			{
				members[component[i]].push_back(i);
			}
			else
			{
				left[component[i]].push_back(i);
				noo++;
			}
		}
		
		double rank[n];
		for(i=0;i<n;i++){
			rank[i]=1.0/n;
		}

		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}

		int thresh=100000;
		
		double initial[n];
		memset(initial,0,sizeof(initial));
		
		for(i=0;i<par.size()-1;i++){
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++)
			{
				int sum=0;
				for(j=0;j<members[order[w]].size();j++){
					sum=sum+rgraph[members[order[w]][j]].size();
				}
				if(sum>thresh)
				{
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
			{
				long long val=computeparallelic(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
			{
				long long val=computerankic(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}

	if(optident==1 && optchain==1 && optdead==1)
	{
		int parent[n];
		vector < vector < int > > left(com);
		for(i=0;i<n;i++){
			parent[i]=i;
		}
		vector < vector <  pair  <  pair < long long , int > , int >  > > hvalues(n);
		
		for(i=0;i<n;i++){
			if(rgraph[i].size()!=1 && rgraph[i].size()!=2) continue;
			if(rgraph[i].size()==1){
				hvalues[(rgraph[i][0])%n].push_back(make_pair(make_pair(rgraph[i][0],component[i]),i));
			}
			else{
				long long val=max(rgraph[i][1]+1,rgraph[i][0]+1)*(long long)(n+1)+min(rgraph[i][0]+1,rgraph[i][1]+1);
				hvalues[(val)%(long long)n ].push_back(make_pair(make_pair(val,component[i]),i));
			}
		}
	
		for(i=0;i<n;i++){
			sort(hvalues[i].begin(),hvalues[i].end());
		}
		
		for(int k=0;k<n;k++){
			for(i=0;i<hvalues[k].size();i++){
				for(j=i;j<hvalues[k].size() && hvalues[k][j].first==hvalues[k][i].first ;j++){
					parent[hvalues[k][j].second]=hvalues[k][i].second;
				}
				i=j-1;
			}
		}
		hvalues.clear();

		int noo=0;
		for(i=0;i<n;i++){
			if(parent[i]==i){
				members[component[i]].push_back(i);
			}
			else{
				left[component[i]].push_back(i);
				noo++;
			}
		}
		
		double rank[n];
		for(i=0;i<n;i++) rank[i]=1.0/n;
		vector < int > par;
		par.push_back(0);
		for(i=0;i<com;i++){
			int j=i;
			while(j<com && nvisit[order[j]]==nvisit[order[i]]){
				j++;
			}
			par.push_back(j);
			i=j-1;
		}
	
		int thresh=100000;
	
		double initial[n];
		memset(initial,0,sizeof(initial));

		for(i=0;i<par.size()-1;i++){
			int pivot=par[i];
			for(int w=par[i];w<par[i+1];w++)
			{
				int sum=0;
				for(j=0;j<members[order[w]].size();j++)
					sum=sum+rgraph[members[order[w]][j]].size();
				if(sum>thresh)
				{
					int temp=order[pivot];
					order[pivot]=order[w];
					order[w]=temp;
					pivot++;
				}
			}
			int k;
			for(w=par[i];w<pivot;w++)
			{
#pragma omp parallel for private(j,k)
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
#pragma omp parallel for private(w,j,k)
			for(w=pivot;w<par[i+1];w++)
			{
				for(j=0;j<members[order[w]].size();j++)
				{
					int node=members[order[w]][j];
					for(k=0;k<rcwgraph[node].size();k++)
						initial[node]+=rank[rcwgraph[node][k]]/outdeg[rcwgraph[node][k]];
					initial[node]=0.85*initial[node];
				}
			}
			for(j=par[i];j<pivot;j++)
			{
				long long val=computeparallelic(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
#pragma omp parallel for private(j)
			for(j=pivot;j<par[i+1];j++)
			{
				long long val=computerankic(rcgraph,parent,left[order[j]],members[order[j]].size(),outdeg,members[order[j]],rank,initial,levelz,redir,powers);
			}
		}
		
		double sum=0;
		for(i=0;i<n;i++){
			sum=sum+rank[i];
		}
		for(i=0;i<n;i++){
			rank[i]=rank[i]/sum;
		}
		for(i=0;i<n;i++){
			fout << rank[i] << "\n";
		}
	}
	return 0;
}

